#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of libcu++, the C++ Standard Library for your entire system,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <thrust/device_vector.h>

#include <cuda/std/__algorithm>
#include <cuda/std/cmath>
#include <cuda/std/ranges>

#include "nvbench_helper.cuh"

template <class T>
struct square_t
{
  __host__ __device__ void operator()(T& x) const noexcept
  {
    x = x * x + cuda::std::sin(static_cast<double>(x));
  }
};

template <typename T>
static void basic(nvbench::state& state, nvbench::type_list<T>)
{
  const auto elements = static_cast<std::size_t>(state.get_int64("Elements"));

  std::vector<T> in(elements, T{1});

  state.add_element_count(elements);
  state.add_global_memory_reads<T>(elements);
  state.add_global_memory_writes<T>(elements);

  state.exec(nvbench::exec_tag::no_batch, [&](nvbench::launch& launch) {
    cuda::std::ranges::for_each(cuda::std::execution::unseq_host, in.begin(), in.end(), square_t<T>{});
  });
}

NVBENCH_BENCH_TYPES(basic, NVBENCH_TYPE_AXES(fundamental_types))
  .set_name("base")
  .set_type_axes_names({"T{ct}"})
  .add_int64_power_of_two_axis("Elements", nvbench::range(14, 18, 4));
