#include "hip/hip_runtime.h"
#include <unittest/unittest.h>

#include <thrust/complex.h>
#include <thrust/detail/config.h>

#include <complex>
#include <iostream>
#include <sstream>
#include <type_traits>

/* 
   The following tests do not check for the numerical accuracy of the operations.
   That is tested in a separate program (complex_accuracy.cpp) which requires mpfr, 
   and takes a lot of time to run.   
 */

template<typename T>
struct TestComplexSizeAndAlignment
{
  void operator()()
  {
    THRUST_STATIC_ASSERT(
      sizeof(thrust::complex<T>) == sizeof(T) * 2
    );
    THRUST_STATIC_ASSERT(
      THRUST_ALIGNOF(thrust::complex<T>) == THRUST_ALIGNOF(T) * 2
    );

    THRUST_STATIC_ASSERT(
      sizeof(thrust::complex<T const>) == sizeof(T) * 2
    );
    THRUST_STATIC_ASSERT(
      THRUST_ALIGNOF(thrust::complex<T const>) == THRUST_ALIGNOF(T) * 2
    );
  }
};
SimpleUnitTest<TestComplexSizeAndAlignment, FloatingPointTypes> TestComplexSizeAndAlignmentInstance;

template<typename T>
struct TestComplexConstructors
{
  void operator()(void)
  {
    thrust::host_vector<T> data = unittest::random_samples<T>(2);
    
    thrust::complex<T> a(data[0],data[1]);
    thrust::complex<T> b(a);
    a = thrust::complex<T>(data[0],data[1]);
    ASSERT_ALMOST_EQUAL(a,b);
    
    a = thrust::complex<T>(data[0]);
    ASSERT_EQUAL(data[0], a.real());
    ASSERT_EQUAL(T(0), a.imag());
    
    a = thrust::complex<T>();
    ASSERT_ALMOST_EQUAL(a,std::complex<T>(0));
    
    a = thrust::complex<T>(thrust::complex<float>(static_cast<float>(data[0]),static_cast<float>(data[1])));
    ASSERT_ALMOST_EQUAL(a,b);
    
    a = thrust::complex<T>(thrust::complex<double>(static_cast<double>(data[0]),static_cast<double>(data[1])));
    ASSERT_ALMOST_EQUAL(a,b);
    
    a = thrust::complex<T>(std::complex<float>(static_cast<float>(data[0]),static_cast<float>(data[1])));
    ASSERT_ALMOST_EQUAL(a,b);
    
    a = thrust::complex<T>(std::complex<double>(static_cast<double>(data[0]),static_cast<double>(data[1])));
    ASSERT_ALMOST_EQUAL(a,b);
  }
};
SimpleUnitTest<TestComplexConstructors, FloatingPointTypes> TestComplexConstructorsInstance;


template<typename T>
struct TestComplexGetters
{
  void operator()(void)
  {
    thrust::host_vector<T> data = unittest::random_samples<T>(2);

    thrust::complex<T> z(data[0], data[1]);

    ASSERT_EQUAL(data[0], z.real());
    ASSERT_EQUAL(data[1], z.imag());

    z.real(data[1]);
    z.imag(data[0]);
    ASSERT_EQUAL(data[1], z.real());
    ASSERT_EQUAL(data[0], z.imag());

    volatile thrust::complex<T> v(data[0], data[1]);

    ASSERT_EQUAL(data[0], v.real());
    ASSERT_EQUAL(data[1], v.imag());

    v.real(data[1]);
    v.imag(data[0]);
    ASSERT_EQUAL(data[1], v.real());
    ASSERT_EQUAL(data[0], v.imag());
  }
};
SimpleUnitTest<TestComplexGetters, FloatingPointTypes> TestComplexGettersInstance;

template<typename T>
struct TestComplexMemberOperators
{
  void operator()(void)
  {
    thrust::host_vector<T> data_a = unittest::random_samples<T>(2);
    thrust::host_vector<T> data_b = unittest::random_samples<T>(2);

    thrust::complex<T> a(data_a[0], data_a[1]);
    thrust::complex<T> b(data_b[0], data_b[1]);

    std::complex<T> c(a);
    std::complex<T> d(b);

    a += b;
    c += d;
    ASSERT_ALMOST_EQUAL(a,c);

    a -= b;
    c -= d;
    ASSERT_ALMOST_EQUAL(a,c);

    a *= b;
    c *= d;
    ASSERT_ALMOST_EQUAL(a,c);

    a /= b;
    c /= d;
    ASSERT_ALMOST_EQUAL(a,c);

    // casting operator
    c = (std::complex<T>)a;
  }
};
SimpleUnitTest<TestComplexMemberOperators, FloatingPointTypes> TestComplexMemberOperatorsInstance;


template<typename T>
struct TestComplexBasicArithmetic
{
  void operator()(void)
  {
    thrust::host_vector<T> data = unittest::random_samples<T>(2);

    thrust::complex<T> a(data[0], data[1]);
    std::complex<T> b(a);

    // Test the basic arithmetic functions against std
    
    ASSERT_ALMOST_EQUAL(abs(a),abs(b));

    ASSERT_ALMOST_EQUAL(arg(a),arg(b));

    ASSERT_ALMOST_EQUAL(norm(a),norm(b));

    ASSERT_EQUAL(conj(a),conj(b));

    ASSERT_ALMOST_EQUAL(thrust::polar(data[0],data[1]),std::polar(data[0],data[1]));

    // random_samples does not seem to produce infinities so proj(z) == z
    ASSERT_EQUAL(proj(a),a);
    
  }
};
SimpleUnitTest<TestComplexBasicArithmetic, FloatingPointTypes> TestComplexBasicArithmeticInstance;


template<typename T>
struct TestComplexBinaryArithmetic
{
  void operator()(void)
  {
    thrust::host_vector<T> data_a = unittest::random_samples<T>(2);
    thrust::host_vector<T> data_b = unittest::random_samples<T>(2);

    thrust::complex<T> a(data_a[0], data_a[1]);
    thrust::complex<T> b(data_b[0], data_b[1]);

    ASSERT_ALMOST_EQUAL(a * b, std::complex<T>(a) * std::complex<T>(b));
    ASSERT_ALMOST_EQUAL(a * data_b[0], std::complex<T>(a) * data_b[0]);
    ASSERT_ALMOST_EQUAL(data_a[0] * b, data_b[0] * std::complex<T>(b));

    ASSERT_ALMOST_EQUAL(a / b, std::complex<T>(a) / std::complex<T>(b));
    ASSERT_ALMOST_EQUAL(a / data_b[0], std::complex<T>(a) / data_b[0]);
    ASSERT_ALMOST_EQUAL(data_a[0] / b, data_b[0] / std::complex<T>(b));

    ASSERT_EQUAL(a + b, std::complex<T>(a) + std::complex<T>(b));
    ASSERT_EQUAL(a + data_b[0], std::complex<T>(a) + data_b[0]);
    ASSERT_EQUAL(data_a[0] + b, data_b[0] + std::complex<T>(b));

    ASSERT_EQUAL(a - b, std::complex<T>(a) - std::complex<T>(b));
    ASSERT_EQUAL(a - data_b[0], std::complex<T>(a) - data_b[0]);
    ASSERT_EQUAL(data_a[0] - b, data_b[0] - std::complex<T>(b));
  }
};
SimpleUnitTest<TestComplexBinaryArithmetic, FloatingPointTypes> TestComplexBinaryArithmeticInstance;




template<typename T>
struct TestComplexUnaryArithmetic
{
  void operator()(void)
  {
    thrust::host_vector<T> data_a = unittest::random_samples<T>(2);

    thrust::complex<T> a(data_a[0], data_a[1]);

    ASSERT_EQUAL(+a,+std::complex<T>(a));
    ASSERT_EQUAL(-a,-std::complex<T>(a));
    
  }
};
SimpleUnitTest<TestComplexUnaryArithmetic, FloatingPointTypes> TestComplexUnaryArithmeticInstance;


template<typename T>
struct TestComplexExponentialFunctions
{
  void operator()(void)
  {
    thrust::host_vector<T> data_a = unittest::random_samples<T>(2);

    thrust::complex<T> a(data_a[0], data_a[1]);
    std::complex<T> b(a);

    ASSERT_ALMOST_EQUAL(exp(a),exp(b));
    ASSERT_ALMOST_EQUAL(log(a),log(b));
    ASSERT_ALMOST_EQUAL(log10(a),log10(b));
    
  }
};
SimpleUnitTest<TestComplexExponentialFunctions, FloatingPointTypes> TestComplexExponentialFunctionsInstance;


template<typename T>
struct TestComplexPowerFunctions
{
  void operator()(void)
  {
    thrust::host_vector<T> data_a = unittest::random_samples<T>(2);
    thrust::host_vector<T> data_b = unittest::random_samples<T>(2);

    thrust::complex<T> a(data_a[0], data_a[1]);
    thrust::complex<T> b(data_b[0], data_b[1]);
    std::complex<T> c(a);
    std::complex<T> d(b);

    ASSERT_ALMOST_EQUAL(pow(a,b),pow(c,d));
    ASSERT_ALMOST_EQUAL(pow(a,b.real()),pow(c,d.real()));
    ASSERT_ALMOST_EQUAL(pow(a.real(),b),pow(c.real(),d));

    ASSERT_ALMOST_EQUAL(sqrt(a),sqrt(c));

  }
};
SimpleUnitTest<TestComplexPowerFunctions, FloatingPointTypes> TestComplexPowerFunctionsInstance;

template<typename T>
struct TestComplexTrigonometricFunctions
{
  void operator()(void)
  {
    thrust::host_vector<T> data_a = unittest::random_samples<T>(2);

    thrust::complex<T> a(data_a[0], data_a[1]);
    std::complex<T> c(a);

    ASSERT_ALMOST_EQUAL(cos(a),cos(c));
    ASSERT_ALMOST_EQUAL(sin(a),sin(c));
    ASSERT_ALMOST_EQUAL(tan(a),tan(c));

    ASSERT_ALMOST_EQUAL(cosh(a),cosh(c));
    ASSERT_ALMOST_EQUAL(sinh(a),sinh(c));
    ASSERT_ALMOST_EQUAL(tanh(a),tanh(c));

#if THRUST_CPP_DIALECT >= 2011

    ASSERT_ALMOST_EQUAL(acos(a),acos(c));
    ASSERT_ALMOST_EQUAL(asin(a),asin(c));
    ASSERT_ALMOST_EQUAL(atan(a),atan(c));

    ASSERT_ALMOST_EQUAL(acosh(a),acosh(c));
    ASSERT_ALMOST_EQUAL(asinh(a),asinh(c));
    ASSERT_ALMOST_EQUAL(atanh(a),atanh(c));

#endif


  }
};
SimpleUnitTest<TestComplexTrigonometricFunctions, FloatingPointTypes> TestComplexTrigonometricFunctionsInstance;

template<typename T>
struct TestComplexStreamOperators
{
  void operator()(void)
  {
    thrust::host_vector<T> data_a = unittest::random_samples<T>(2);
    thrust::complex<T> a(data_a[0], data_a[1]);
    std::stringstream out;
    out << a;
    thrust::complex<T> b;
    out >> b;
    ASSERT_ALMOST_EQUAL(a,b);
  }
};
SimpleUnitTest<TestComplexStreamOperators, FloatingPointTypes> TestComplexStreamOperatorsInstance;

#if THRUST_CPP_DIALECT >= 2011
template<typename T>
struct TestComplexStdComplexDeviceInterop
{
  void operator()()
  {
    thrust::host_vector<T> data = unittest::random_samples<T>(6);
    std::vector<std::complex<T> > vec(10);
    vec[0] = std::complex<T>(data[0], data[1]);
    vec[1] = std::complex<T>(data[2], data[3]);
    vec[2] = std::complex<T>(data[4], data[5]);

    thrust::device_vector<thrust::complex<T> > device_vec = vec;
    ASSERT_ALMOST_EQUAL(vec[0].real(), thrust::complex<T>(device_vec[0]).real());
    ASSERT_ALMOST_EQUAL(vec[0].imag(), thrust::complex<T>(device_vec[0]).imag());
    ASSERT_ALMOST_EQUAL(vec[1].real(), thrust::complex<T>(device_vec[1]).real());
    ASSERT_ALMOST_EQUAL(vec[1].imag(), thrust::complex<T>(device_vec[1]).imag());
    ASSERT_ALMOST_EQUAL(vec[2].real(), thrust::complex<T>(device_vec[2]).real());
    ASSERT_ALMOST_EQUAL(vec[2].imag(), thrust::complex<T>(device_vec[2]).imag());
  }
};
SimpleUnitTest<TestComplexStdComplexDeviceInterop, FloatingPointTypes> TestComplexStdComplexDeviceInteropInstance;
#endif


template<typename TypeList>
struct TestComplexAllMembersWithPromoting
{
  void operator()(void)
  {
    typedef unittest::get_type_t<TypeList,0> T1;
    typedef unittest::get_type_t<TypeList,1> T2;

    thrust::host_vector<T1> data_a = unittest::random_samples<T1>(2);
    thrust::host_vector<T2> data_b = unittest::random_samples<T2>(4);

    const T1 a = data_a[0];
    const T1 b = data_a[1];
    const T2 c = data_b[2];
    const T2 d = data_b[3];

    const thrust::complex<T1> elem(a, b);
    const std::complex<T1> std_elem = elem;

    ASSERT_ALMOST_EQUAL(elem, elem);
    ASSERT_ALMOST_EQUAL(std_elem, std_elem);

    ASSERT_EQUAL(elem.real(), a);
    ASSERT_EQUAL(elem.imag(), b);

    // copy ctor
    {
      thrust::complex<T2> elem2(elem);
      ASSERT_EQUAL(elem2.real(), (T2) a);
      ASSERT_EQUAL(elem2.imag(), (T2) b);
    }

    // copy ctor with std::complex
    {
      thrust::complex<T2> elem2(std_elem);
      ASSERT_EQUAL(elem2.real(), (T2) a);
      ASSERT_EQUAL(elem2.imag(), (T2) b);
    }

    // assignment from real number
    {
      thrust::complex<T2> elem2 = a;
      ASSERT_EQUAL(elem2.real(), (T2) a);
      ASSERT_EQUAL(elem2.imag(), 0.0F);
    }
    
    // asignment with std::complex;
    {
      thrust::complex<T1> elem2 = std_elem;
      ASSERT_EQUAL(elem2, std_elem);
    }

    // asignment with std::complex, other T
    {
      thrust::complex<T2> elem2 = std_elem;
      ASSERT_EQUAL(elem2, std_elem);
    }

    // no conversion from complex<float> to complex<double>, rest works
    //{
    //  thrust::complex<T2> elem2 = elem;
    //  ASSERT_EQUAL(elem2.real(), a);
    //  ASSERT_EQUAL(elem2.imag(), b);
    //}

    // assignment add
    {
      thrust::complex<T2> elem2(c, d);
      elem2 += elem;
      ASSERT_EQUAL(elem2.real(), a+c);
      ASSERT_EQUAL(elem2.imag(), b+d);
    }

    // assignment substraction
    {
      thrust::complex<T2> elem2(c, d);
      elem2 -= elem;
      ASSERT_EQUAL(elem2.real(), c-a);
      ASSERT_EQUAL(elem2.imag(), d-b);
    }

    // assignment multiplication
    {
      thrust::complex<T2> elem2(c, d);
      elem2 *= elem;
      ASSERT_ALMOST_EQUAL(elem2.real(), (a*c - b*d));
      ASSERT_ALMOST_EQUAL(elem2.imag(), (a*d + b*c));
    }

    // assignment division
    {
      thrust::complex<T2> elem2(c, d);
      elem2 /= elem;
      ASSERT_ALMOST_EQUAL(elem2.real(), (a*c + b*d)/(a*a + b*b));
      ASSERT_ALMOST_EQUAL(elem2.imag(), (a*d - b*c)/(a*a + b*b));
    }

    // assignment add with real
    {
      thrust::complex<T2> elem2(c, d);
      elem2 += a;
      ASSERT_EQUAL(elem2.real(), c+a);
    }

    // assignment substraction with real
    {
      thrust::complex<T2> elem2(c, d);
      elem2 -= a;
      ASSERT_EQUAL(elem2.real(), c-a);
    }

    // assignment multiplication with real
    {
      thrust::complex<T2> elem2(c, d);
      elem2 *= a;
      ASSERT_ALMOST_EQUAL(elem2.real(), (a*c));
      ASSERT_ALMOST_EQUAL(elem2.imag(), (a*d));
    }

    // assignment division with real
    {
      thrust::complex<T2> elem2(c, d);
      elem2 /= a;
      ASSERT_ALMOST_EQUAL(elem2.real(), (a*c)/(a*a));
      ASSERT_ALMOST_EQUAL(elem2.imag(), (a*d)/(a*a));
    }

    // use .real() and .imag() to change real and imag
    {
      thrust::complex<T1> elem2(T1{0.0}, T1{0.0});
      elem2.real(static_cast<T1>(a));
      ASSERT_EQUAL(elem2.real(), static_cast<T1>(a));

      elem2.imag(static_cast<T1>(b));
      ASSERT_EQUAL(elem2.imag(), static_cast<T1>(b));
    }

    // comparision operators
    ASSERT_EQUAL(thrust::complex<T1>(a, b) == thrust::complex<T1>(a, b), true);
    ASSERT_EQUAL(thrust::complex<T1>(a, b) == thrust::complex<T2>(a, b), true);
    ASSERT_EQUAL(thrust::complex<T1>(a, 0.0) == a, true);
    //ASSERT_EQUAL(thrust::complex<T1>(a, 0.0) == static_cast<T2>(a), true);
    ASSERT_EQUAL(a == thrust::complex<T1>(a, 0.0), true);
    //ASSERT_EQUAL(static_cast<T2>(a) == thrust::complex<T1>(a, 0), true);
    ASSERT_EQUAL(std::complex<T1>(a, b) == thrust::complex<T2>(a, b), true);
    ASSERT_EQUAL(thrust::complex<T1>(a, b) == std::complex<T2>(a, b), true);


    ASSERT_EQUAL(thrust::complex<T1>(a, b) != thrust::complex<T1>(c, d), true);
    ASSERT_EQUAL(thrust::complex<T1>(a, b) != thrust::complex<T2>(c, d), true);
    ASSERT_EQUAL(thrust::complex<T1>(a, 0) != b, true);
    //ASSERT_EQUAL(thrust::complex<T1>(a, 0) != c, true);
    ASSERT_EQUAL(b != thrust::complex<T1>(a, 0), true);
    //ASSERT_EQUAL(c != thrust::complex<T1>(a, 0), true);
    ASSERT_EQUAL(std::complex<T1>(a, b) != thrust::complex<T2>(c, d), true);
    ASSERT_EQUAL(thrust::complex<T1>(a, b) != std::complex<T2>(c, d), true);

    // abs
    ASSERT_ALMOST_EQUAL(thrust::abs(elem), std::abs(std_elem));

    // arg
    ASSERT_ALMOST_EQUAL(thrust::arg(elem), std::arg(std_elem));

    // norm
    ASSERT_ALMOST_EQUAL(thrust::norm(elem), std::norm(std_elem));

    // conj
    ASSERT_ALMOST_EQUAL(thrust::conj(elem), std::conj(std_elem));

    // polar, does not compile, cos/sin/... missing
    ASSERT_ALMOST_EQUAL(thrust::polar(a), std::polar(a));
    ASSERT_ALMOST_EQUAL(thrust::polar(a, b), std::polar(a, b));
    //ASSERT_ALMOST_EQUAL(thrust::polar(a, c), std::polar(a, c));

    // proj
    ASSERT_ALMOST_EQUAL(thrust::proj(elem), std::proj(std_elem));

    // add
    auto result = thrust::complex<T1>(a, b) + thrust::complex<T2>(c, d);
    ASSERT_EQUAL(result.real(), a + c);
    ASSERT_EQUAL(result.imag(), b + d); 

    // add with real, does not compile
    //cresult = thrust::complex<T1>(a, b) + c;
    //ASSERT_EQUAL(cresult.real(), a + c);
    //ASSERT_EQUAL(cresult.imag(), b); 

    //cresult = a + thrust::complex<T2>(c, d);
    //ASSERT_EQUAL(cresult.real(), a + c);
    //ASSERT_EQUAL(cresult.imag(), d); 

    // substraction
    result = thrust::complex<T1>(a, b) - thrust::complex<T2>(c, d);
    ASSERT_EQUAL(result.real(), a - c);
    ASSERT_EQUAL(result.imag(), b - d);

    // substraction with real, does not compile
    //cresult = thrust::complex<T1>(a, b) - c;
    //ASSERT_EQUAL(cresult.real(), a - c);
    //ASSERT_EQUAL(cresult.imag(), b); 

    //cresult = a - thrust::complex<T2>(c, d);
    //ASSERT_EQUAL(cresult.real(), a - c);
    //ASSERT_EQUAL(cresult.imag(), d); 

    // multiplication, does not compile for promotion case
    result = thrust::complex<T1>(a, b) * thrust::complex<T2>(c, d);
    ASSERT_ALMOST_EQUAL(result.real(), (a*c - b*d));
    ASSERT_ALMOST_EQUAL(result.imag(), (a*d + b*c));

    // multiplication with real, does not compile
    //cresult = thrust::complex<T1>(a, b) * c;
    //ASSERT_ALMOST_EQUAL(cresult.real(), (a*c));
    //ASSERT_ALMOST_EQUAL(cresult.imag(), (b*c));

    //cresult = a * thrust::complex<T2>(c, d);
    //ASSERT_ALMOST_EQUAL(cresult.real(), (a*c));
    //ASSERT_ALMOST_EQUAL(cresult.imag(), (a*d));

    // division, does not compile for promotion case
    result = thrust::complex<T1>(a, b) / thrust::complex<T2>(c, d);
    ASSERT_ALMOST_EQUAL(result.real(), (a*c + b*d)/(c*c + d*d));
    ASSERT_ALMOST_EQUAL(result.imag(), (b*c - a*d)/(c*c + d*d));

    // division with real, does not compile
    //cresult = thrust::complex<T1>(a, b) / c;
    //ASSERT_ALMOST_EQUAL(elem2.real(), (a*c)/(c*c + d*d));
    //ASSERT_ALMOST_EQUAL(elem2.imag(), (b*c)/(c*c + d*d));

    //cresult = a / thrust::complex<T2>(c, d);
    //ASSERT_ALMOST_EQUAL(elem2.real(), (a*c)/(c*c));
    //ASSERT_ALMOST_EQUAL(elem2.imag(), (-a*d)/(c*c));

    // unary+
    ASSERT_EQUAL(+elem, elem);

    // unary-
    ASSERT_EQUAL(-elem, elem * thrust::complex<T1>(T1{-1.0}, T1{0.0}));

    // exp
    ASSERT_ALMOST_EQUAL(thrust::exp(elem), std::exp(std_elem));

    // log
    ASSERT_ALMOST_EQUAL(thrust::log(elem), std::log(std_elem));

    // log10
    ASSERT_ALMOST_EQUAL(thrust::log10(elem), std::log10(std_elem));

    // pow
    ASSERT_ALMOST_EQUAL(thrust::pow(elem, thrust::complex<T1>(a, b)), std::pow(std_elem, std::complex<T1>(a, b)));
    ASSERT_ALMOST_EQUAL(thrust::pow(elem, thrust::complex<T2>(c, d)), std::pow(std_elem, std::complex<T2>(c, d)));
    ASSERT_ALMOST_EQUAL(thrust::pow(thrust::complex<T2>(c, d), elem), std::pow(std::complex<T2>(c, d), std_elem));

    // pow with reals
    ASSERT_ALMOST_EQUAL(thrust::pow(elem, T1{a}), std::pow(std_elem, T1{a}));
    ASSERT_ALMOST_EQUAL(thrust::pow(T1{a}, elem), std::pow(T1{a}, std_elem));
    ASSERT_ALMOST_EQUAL(thrust::pow(elem, T2{c}), std::pow(std_elem, T2{c}));
    ASSERT_ALMOST_EQUAL(thrust::pow(T2{c}, elem), std::pow(T2{c}, std_elem));

    // srqt
    ASSERT_ALMOST_EQUAL(thrust::sqrt(elem), std::sqrt(std_elem));

    // cos
    ASSERT_ALMOST_EQUAL(thrust::cos(elem), std::cos(std_elem));

    // sin
    ASSERT_ALMOST_EQUAL(thrust::sin(elem), std::sin(std_elem));

    // tan
    ASSERT_ALMOST_EQUAL(thrust::tan(elem), std::tan(std_elem));

    // cosh
    ASSERT_ALMOST_EQUAL(thrust::cosh(elem), std::cosh(std_elem));

    // sinh
    ASSERT_ALMOST_EQUAL(thrust::sinh(elem), std::sinh(std_elem));

    // tanh
    ASSERT_ALMOST_EQUAL(thrust::tanh(elem), std::tanh(std_elem));

    // acos
    ASSERT_ALMOST_EQUAL(thrust::acos(elem), std::acos(std_elem));

    // atan
    ASSERT_ALMOST_EQUAL(thrust::atan(elem), std::atan(std_elem));

    // atanh
    ASSERT_ALMOST_EQUAL(thrust::atanh(elem), std::atanh(std_elem));

  }
};


SimpleUnitTest<TestComplexAllMembersWithPromoting,
               unittest::type_list<unittest::type_list<double, double>,
                                   unittest::type_list<float, float>,
                                   unittest::type_list<float, double>,
                                   unittest::type_list<double, float>>>
  testComplexAllMembersWithPromotingInstance;